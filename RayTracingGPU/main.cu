#include "hip/hip_runtime.h"
#include <cstdint>

#define STB_IMAGE_WRITE_IMPLEMENTATION
#include "stb_image_write.h"

#include "rtutil.cuh"

#include "camera.cuh"
#include "hitbox.cuh"
#include "hitbox_list.cuh"
#include "material.cuh"
#include "sphere.cuh"

__global__ void render(double* img, camera** cam, hitbox_list** world, int image_width, hiprandState *states, double frac, uint8_t* imgu) 
{
    int x = blockIdx.x%image_width;
    int y = blockIdx.y;
    int pixel_count = gridDim.y*image_width;
    // int image_width = gridDim.x;
    // int y = threadIdx.x;
    int b_id = image_width*y + x;
    hiprandState localState = states[(b_id+threadIdx.x)%pixel_count];

    color pixel_color = (*cam)->render(*world, x, y, localState);

    states[(b_id+threadIdx.x)%pixel_count] = localState;

    int pos = y*image_width*3 + x*3;

    atomicAdd(&img[pos], frac*pixel_color.x());
    atomicAdd(&img[pos+1], frac*pixel_color.y());
    atomicAdd(&img[pos+2], frac*pixel_color.z());

    // img[pos] = frac*pixel_color.x();
    // img[pos+1] = frac*pixel_color.y();
    // img[pos+2] = frac*pixel_color.z();

    // write_color(imgu, img, y, x, image_width);
    // write_color(pixel_color, imgu, y, x, image_width);
}

__global__ void make_image(double* img, uint8_t* imgu, int image_width) 
{
    int x = blockIdx.x;
    int y = blockIdx.y; 

    write_color(imgu, img, y, x, image_width);
}

//Cuda functions

__global__ void setup_random(hiprandState *states)
{
    unsigned long long id = gridDim.x*blockIdx.y + blockIdx.x;
    hiprand_init(id, 0, 0, &(states[id]));
    
    // printf("\n(%d, %d):\n",blockIdx.y,blockIdx.x);
    // for(int i = 0; i < 5; i++)
    // {
    //     hiprandState st = states[id];
    //     printf("%lf\n",hiprand_uniform_double(&st));
    // }
}

__global__ void createCam(camera** cam, double aspectRatio, int imgWidth, double vFov, point3 lookfrom, point3 lookat, vec3 vup)
{
    (*cam) = new camera();
    (*cam)->init(aspectRatio, imgWidth, vFov, lookfrom, lookat, vup);
}

__global__ void initWorld(hitbox_list** world)
{
    (*world) = new hitbox_list();
}

__global__ void initMatLambertian(material** mat, color attenuation)
{
    (*mat) = new lambertian(attenuation);
}

__global__ void initMatMetal(material** mat, color attenuation, double fuzz = 0)
{
    (*mat) = new metal(attenuation, fuzz);
}

__global__ void initMatDielectric(material** mat, double index)
{
    (*mat) = new dielectric(index);
}

__global__ void addSphere(hitbox_list** world, point3 center, double radius, material** mat)
{
    (*world)->add(new sphere(center, radius, *mat));
}

__global__ void clean(camera** cam, hitbox_list** world)
{
    delete (*cam);
    (*world)->clear();
    delete (*world);
}

__global__ void delMat(material** mat)
{
    delete (*mat);
}

int main(void)
{
    init_constants();

    // Image

    auto aspect_ratio = 16.0 / 9.0;
    unsigned int image_width = 1920;
    // unsigned int image_width = 4;

    // Calculate the image height, and ensure that it's at least 1.
    unsigned int image_height = max(1,int(image_width / aspect_ratio));

    // World

    hitbox_list** world;
    hipMalloc(&world, sizeof(hitbox_list*));

    material **material_ground, **material_center, **material_left, **material_bubble, **material_right;
    hipMalloc(&material_ground, sizeof(material*));
    hipMalloc(&material_center, sizeof(material*));
    hipMalloc(&material_left, sizeof(material*));
    hipMalloc(&material_bubble, sizeof(material*));
    hipMalloc(&material_right, sizeof(material*));
    initMatLambertian<<<1,1>>>(material_ground, color(0.8, 0.8, 0.0));
    initMatLambertian<<<1,1>>>(material_center, color(0.1, 0.2, 0.5));
    initMatDielectric<<<1,1>>>(material_left, 1.50);
    initMatDielectric<<<1,1>>>(material_bubble, 1.00/1.50);
    initMatMetal<<<1,1>>>(material_right, color(0.8, 0.6, 0.2), 1.0);

    initWorld<<<1,1>>>(world);
    addSphere<<<1,1>>>(world, point3(0.0, -100.5, -1.0), 100.0, material_ground);
    addSphere<<<1,1>>>(world, point3(0.0, 0.0, -1.2), 0.5, material_center);
    addSphere<<<1,1>>>(world, point3(-1.0, 0.0, -1.0), 0.5, material_left);
    addSphere<<<1,1>>>(world, point3(-1.0, 0.0, -1.0), 0.4, material_bubble);
    addSphere<<<1,1>>>(world, point3(1.0, 0.0, -1.0), 0.5, material_right);


    // Camera

    double vFov     = 30.0;
    point3 lookfrom = point3(-2,2,1);
    point3 lookat   = point3(0,0,-1);
    vec3   vup      = vec3(0,1,0);

    camera** cam;
    hipMalloc(&cam, sizeof(camera*));
    createCam<<<1,1>>>(cam, aspect_ratio, image_width, vFov, lookfrom, lookat, vup);

    // Render

    unsigned int samples = 2048;
    // unsigned int divisions = samples/512;
    unsigned int divisions = (samples+511)/512;
    // unsigned int rem = samples-divisions*512;
    unsigned int samples_per_block = samples/divisions;
    double frac = 1.0/(samples_per_block*divisions);
    // frac = 1.0;

    std::cout << "Number of samples are "<< samples_per_block*divisions <<".\n";

    uint8_t* img;
    uint8_t* img_device;
    double* img_doubles;
    hiprandState* rand_states;


    img = (uint8_t*) malloc(sizeof(uint8_t)*image_width*image_height*3);
    hipMalloc(&img_device, image_width*image_height*3);
    hipMalloc(&img_doubles, image_width*image_height*3*sizeof(double));
    hipMemset(img_doubles, 0.0, image_width*image_height*3);
    hipMalloc(&rand_states, image_width*image_height*sizeof(hiprandState));

    setup_random<<<{image_width, image_height, 1}, 1>>>(rand_states);

    hipDeviceSynchronize();

    // for (int i = 0; i < divisions; i++)
    // {
    //     render<<<{image_width, image_height, 1}, 512>>>(img_doubles, cam, world, image_width, rand_states, frac, img_device);
    // }
    // render<<<{image_width, image_height, 1}, rem>>>(img_doubles, cam, world, image_width, rand_states, frac, img_device);

    render<<<{image_width, image_height, divisions}, samples_per_block>>>(img_doubles, cam, world, image_width, rand_states, frac, img_device);

    hipDeviceSynchronize();

    make_image<<<{image_width, image_height, 1}, 1>>>(img_doubles, img_device, image_width);

    hipMemcpy(img, img_device, image_width*image_height*3, hipMemcpyDeviceToHost);

    stbi_write_png("../../Image Samples/envTest.png", image_width, image_height, 3, img, image_width*3);

    clean<<<1,1>>>(cam, world);
    hipFree(world);
    hipFree(cam);
    free(img);
    hipFree(img_device);
    hipFree(img_doubles);
    hipFree(rand_states);
    //TODO: delete materials
    
    return 0;
}