#include "hip/hip_runtime.h"

#include <iostream>
#include <cstdint>

#define STB_IMAGE_WRITE_IMPLEMENTATION
#include "stb_image_write.h"

#include "vec3.cuh"
#include "color.cuh"
#include "ray.cuh"

__device__ bool hit_sphere(const point3& center, double radius, const ray& r) {
    vec3 oc = center - r.origin();
    auto a = dot(r.direction(), r.direction());
    auto b = -2.0 * dot(r.direction(), oc);
    auto c = dot(oc, oc) - radius*radius;
    auto discriminant = b*b - 4*a*c;
    return (discriminant >= 0);
}

__device__ color ray_color(const ray& r) {
    if (hit_sphere(point3(0,0,-1), 0.5, r))
        return color(1, 0, 0);
        
    vec3 unit_direction = unit_vector(r.direction());
    auto a = 0.5*(unit_direction.y() + 1.0);
    return (1.0-a)*color(1.0, 1.0, 1.0) + a*color(0.5, 0.7, 1.0);
}

__global__ void colourImage(uint8_t* img, const vec3 pixel00_loc, const vec3 pixel_delta_u, const vec3 pixel_delta_v, const vec3 camera_center) 
{
    int x = blockIdx.x;
    int y = blockIdx.y;
    int image_width = gridDim.x;
    // int y = threadIdx.x;

    auto pixel_center = pixel00_loc + (x * pixel_delta_u) + (y * pixel_delta_v);
    auto ray_direction = pixel_center - camera_center;
    ray r(camera_center, ray_direction);

    color pixel_color = ray_color(r);
    write_color(pixel_color, img, y, x, image_width);
}

int main(void)
{
    // Image

    auto aspect_ratio = 16.0 / 9.0;
    int image_width = 1280;

    // Calculate the image height, and ensure that it's at least 1.
    int image_height = max(1,int(image_width / aspect_ratio));

    // Camera

    auto focal_length = 1.0;
    auto viewport_height = 2.0;
    auto viewport_width = viewport_height * (double(image_width)/image_height);
    auto camera_center = point3(0, 0, 0);

    // Calculate the vectors across the horizontal and down the vertical viewport edges.
    auto viewport_u = vec3(viewport_width, 0, 0);
    auto viewport_v = vec3(0, -viewport_height, 0);

    // Calculate the horizontal and vertical delta vectors from pixel to pixel.
    auto pixel_delta_u = viewport_u / image_width;
    auto pixel_delta_v = viewport_v / image_height;

    // Calculate the location of the upper left pixel.
    auto viewport_upper_left = camera_center - vec3(0, 0, focal_length) - viewport_u/2 - viewport_v/2;
    auto pixel00_loc = viewport_upper_left + 0.5 * (pixel_delta_u + pixel_delta_v);

    // Render

    uint8_t* img;
    uint8_t* img_device;

    img = (uint8_t*) malloc(sizeof(uint8_t)*image_width*image_height*3);
    hipMalloc(&img_device, image_width*image_height*3);

    colourImage<<<{(unsigned int)image_width, (unsigned int)image_height, 1}, 1>>>(img_device, pixel00_loc, pixel_delta_u, pixel_delta_v, camera_center);

    hipMemcpy(img, img_device, image_width*image_height*3, hipMemcpyDeviceToHost);

    stbi_write_png("../../Image Samples/envTest.png", image_width, image_height, 3, img, image_width*3);
    
    return 0;
}